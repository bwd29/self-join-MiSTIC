#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>



#include "include/tree.cuh"
#include "include/utils.cuh"
#include "include/kernel.cuh"


int main(int argc, char*argv[]){
    


    //reading in command line arguments
	char *filename = argv[1];
	int dim = atoi(argv[2]);
	int tpp = 8;
	int concurent_streams = 2;
	double epsilon;
	sscanf(argv[3], "%lf", &epsilon);

	double time0 = omp_get_wtime();

	std::ifstream file(	filename, std::ios::in | std::ios::binary);
	file.seekg(0, std::ios::end); 
	size_t size = file.tellg();  
	file.seekg(0, std::ios::beg); 
	char * read_buffer = new char[size];
	file.read(read_buffer, size*sizeof(double));
	file.close();

	double time00 = omp_get_wtime();
	printf("\nTime to read in file: %f\n", time00-time0);

	double* A = (double*)read_buffer;//reinterpret as doubles

	int numPoints = size/sizeof(double)/dim;


	printf("\nNumber points: %d ", numPoints);
	printf("\nNumber Dimensions: %d ", dim);
	printf("\nNumber Threads Per Point: %d ", tpp);
	printf("\nNumber Concurent Streams: %d", concurent_streams);
	printf("\nDistance Threshold: %f \n*********************************\n\n", epsilon);




	int *dimension_order = (int*)malloc(sizeof(int)*dim);
	double * dim_ordered_data = (double*)malloc(sizeof(double)*numPoints*dim);

    dimension_order = stddev(A, dim, numPoints);
    #pragma omp parallel for
    for(int i = 0; i < numPoints; i++){
        for(int j = 0; j < dim; j++){
            dim_ordered_data[i*dim + j] = A[i*dim + dimension_order[j]];
        }
    }
    A = dim_ordered_data;;



	double time1 = omp_get_wtime();


	//build tree
	int * pointArray = (int*)malloc(sizeof(int)*numPoints);
	for (int i = 0; i < numPoints; i++){
		point_array[i] = i;
	}


	int ** pointBinNumbers;

	unsigned int binSizes[MAXRP];
	unsigned int binAmounts[MAXRP];
	int maxBinAmount = MAX_BIN;
	int ** tree;
	int numLayers = buildTree(
					&tree,
					A,
					dim,
					numPoints,
					epsilon,
					maxBinAmount,
					pointArray,
					&pointBinNumbers,
					binSizes,
					binAmounts);


    double * point_ordered_data = (double *)malloc(sizeof(double)*numPoints*dim);
    #pragma omp parallel for
	for(int i = 0; i < numPoints; i++){
		for(int j = 0; j < dim; j++){
			point_ordered_data[i*dim+j] = A[pointArray[i]*dim+j];
		}
	}
	A = point_ordered_data;


    int * addIndexes;
    int ** rangeIndexes;
    int ** rangeSizes;
    int * numValidRanges;
    int * calcPerAdd;
    int nonEmptyBins = generateRanges(tree, numPoints, pointArray, pointBinNumbers, numLayers, binSizes, binAmounts, &addIndexes, &rangeIndexes, &rangeSizes, &numValidRanges, &calcPerAdd);

    long long sumCalcs = 0;
    long long sumAdds = 0;
    for(int i = 0; i < nonEmptyBins; i++){
        sumCalcs += calcPerAdd[i];
        sumAdds += numValidRanges[i];
    }

    printf("Number non-empty bins: %d\nNumber of calcs: %ld\nNumber Address for calcs: %ld\n", nonEmptyBins, sumCalcs, sumAdds);

    double time2 = omp_get_wtime();

    printf("Time to build tree: %f\n", time2-time1);

    return 1;

}