#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>



#include "include/tree.cuh"
#include "include/utils.cuh"
#include "include/kernel.cuh"


int main(int argc, char*argv[]){
    


    //reading in command line arguments
	char *filename = argv[1]; // first argument is the file with the dataset as a .bin
	int dim = atoi(argv[2]); // second argument is the dimensionality of the data, i.e. number of columns
	int concurent_streams = 2; // number of cuda streams, should only ever need to be 2 but can be set to a parameter
	double epsilon;
	sscanf(argv[3], "%lf", &epsilon); // third argumernt is the distance threshold being searched

	double time0 = omp_get_wtime(); //start initial timer

	//read in file from binary, only works with doubles if file saved as doubles
	std::ifstream file(	filename, std::ios::in | std::ios::binary);
	file.seekg(0, std::ios::end); 
	size_t size = file.tellg();  
	file.seekg(0, std::ios::beg); 
	char * read_buffer = (char*)malloc(sizeof(char)*size);
	file.read(read_buffer, size*sizeof(double));
	file.close();


	double time00 = omp_get_wtime();
	printf("\nTime to read in file: %f\n", time00-time0);

	double* A = (double*)read_buffer;//reinterpret as doubles

	int numPoints = size/sizeof(double)/dim; // calculate number of points based on the siez of the input

	// can set a subset of the data for easier debugging
	//////////////
	numPoints = 10000;
	////////////

	printf("\nNumber points: %d ", numPoints);
	printf("\nNumber Dimensions: %d ", dim);
	printf("\nNumber Concurent Streams: %d", concurent_streams);
	printf("\nDistance Threshold: %f \n*********************************\n\n", epsilon);

	//if using a small datset for debugging, also run brute force so we can double check results
	if(numPoints <= 10000) 	brute_force( numPoints, dim, epsilon, A);

	double time1 = omp_get_wtime();

	//dimensionOrder holds the order of dimensions sorted by thier varience
	int *dimensionOrder = (int*)malloc(sizeof(int)*dim);

	//dimOrderedData holds the dataset after it has been reordered based on dimensional varience
	double * dimOrderedData = (double*)malloc(sizeof(double)*numPoints*dim);

    dimensionOrder = stddev(A, dim, numPoints); //find the order of dimensions by varience

	// reorder the origional data into the dimesionaly ordered data
	// this makes earlier columns of the data have higher varience than later columns
	// the points maintain thier order relative to other points
	// this can increase short circuiting because higher variences are calculated earlier
    #pragma omp parallel for
    for(int i = 0; i < numPoints; i++){
        for(int j = 0; j < dim; j++){
            dimOrderedData[i*dim + j] = A[i*dim + dimensionOrder[j]];
        }
    }



	// allocate and set an array to keep the order of the points
	// this allows us to refer to the row of the intial data when returning pairs
	int * pointArray = (int*)malloc(sizeof(int)*numPoints);
	for (int i = 0; i < numPoints; i++){
		pointArray[i] = i;
	}


	// poinmt bin numbers holds the bins relative to reference points that each point is in
	int ** pointBinNumbers;

	// binSizes is the number of bins for each layer of the tree , which includes the spread from the previous layer
	unsigned int * binSizes = (unsigned int*)malloc(sizeof(unsigned int)*MAXRP);

	//bin amounts is the number of bins for that reference point, i.e. the range of points / epsilon
	unsigned int * binAmounts = (unsigned int*)malloc(sizeof(unsigned int)*MAXRP);

	//maxBinAmount limits the number of bins that can be in a layer to reduce space complexity, not usually an issue
	int maxBinAmount = MAX_BIN;

	//this will be the tree structure of pointers to layers
	int ** tree;

	//build the tree into tree and returns the number of layers that was selected for the tree
	int numLayers = buildTree(
					&tree, // outputs into this pointer
					dimOrderedData, //uses the data after ordered for dimensions
					dim, // the dimensionality of the data
					numPoints, // the number of points in the dataset
					epsilon, // the distance threshold being searched
					maxBinAmount, // the limmiter on tree width in number of bins
					pointArray, // the ordered points, this will be rearanged when building the tree
					&pointBinNumbers, // this will hold of the bin number for each point relative to each reference point
					binSizes, // the width for each layer of the tree which is built in the fuinction
					binAmounts); //the number of bins for each reference point as in the range / epsilon


	// allocate a data array for used with distance calcs
	// the data is moved around so that point in bin are near eachother in the array
	// the location is based  on the point array that was altered during tree construction
	// data can be organized 2 ways:
	// 1. if DATANORM = true
	//    the data is organized so that the the dimensions of each point are next to eachother
	//	  this allows for coalesced memory accsess on the gpu to increase perfomance
	//
	// 2. if DATANORM is false
	//	  this is the standard stride that was used after dimensional ordering
    double * data = (double *)malloc(sizeof(double)*numPoints*dim);
    #pragma omp parallel for
	for(int i = 0; i < numPoints; i++){
		for(int j = 0; j < dim; j++){
			#if DATANORM
			data[i+numPoints*j] = dimOrderedData[pointArray[i]*dim+j];
			#else
			data[i*dim+j] = dimOrderedData[pointArray[i]*dim+j];
			#endif
		}
	}


	// checking that the last bin size is not negative or zero and that the tree has every data point in it
	printf("Last Layer Bin Count: %d\nTree Check: %d\n",binSizes[numLayers-1], tree[numLayers-1][binSizes[numLayers-1]-1]);

	double time2 = omp_get_wtime();

    printf("Time to build tree: %f\n", time2-time1);


	// addIndexes holds the return from generating ranges
    int * addIndexes;

	// rangeIndexes holds the return from generating ranges that 
    int ** rangeIndexes;
    unsigned int ** rangeSizes;
    int * numValidRanges;
    unsigned long long * calcPerAdd;
	unsigned int *numPointsInAdd;
    int nonEmptyBins = generateRanges(tree,
									  numPoints,
									  pointBinNumbers,
									  numLayers,
									  binSizes,
									  binAmounts,
									  &addIndexes,
									  &rangeIndexes,
									  &rangeSizes,
									  &numValidRanges,
									  &calcPerAdd,
									  &numPointsInAdd);

    unsigned long long sumCalcs = 0;
    unsigned long long sumAdds = 0;
    for(int i = 0; i < nonEmptyBins; i++){
        sumCalcs += calcPerAdd[i];
        sumAdds += numValidRanges[i];
    }

	int * addIndexRange = (int*)malloc(sizeof(int)*nonEmptyBins);
	for(int i = 0; i < nonEmptyBins; i++){
		addIndexRange[i] = tree[numLayers-1][addIndexes[i]];
		// printf("%d\n", addIndexRange[i]);
	}

	unsigned int numSearches = pow(3, numLayers);
	int * linearRangeIndexes = (int*)malloc(sizeof(int)*nonEmptyBins*numSearches);
	unsigned int * linearRangeSizes = (unsigned int*)malloc(sizeof(unsigned int)*nonEmptyBins*numSearches);
	for(int i = 0; i < nonEmptyBins; i++){
		for(int j = 0; j < numValidRanges[i];j++){
			linearRangeIndexes[i*numSearches + j] = tree[numLayers-1][rangeIndexes[i][j]];
			linearRangeSizes[i*numSearches + j] = rangeSizes[i][j];
		}
	}

    printf("Number non-empty bins: %d\nNumber of calcs: %llu\nNumber Address for calcs: %llu\n", nonEmptyBins, sumCalcs, sumAdds);


	double time3 = omp_get_wtime();

	printf("Tree search time: %f\n", time3-time2);


	launchKernel(numLayers, 
				data, 
				dim,
				numPoints,
				epsilon,
				addIndexes,
			    addIndexRange,
				pointArray,
				rangeIndexes,
				rangeSizes,
				numValidRanges,
				numPointsInAdd,
				calcPerAdd,
				nonEmptyBins,
				sumCalcs,
				sumAdds,
				linearRangeIndexes,
				linearRangeSizes);



	double time4 = omp_get_wtime();

	printf("Kernel time: %f\n", time4-time3);

	printf("Total Time: %f\n",time4-time1);









// just freeing memory here
//////////////////////////////////////////////////////////////////////

	for(int i = 0; i < nonEmptyBins; i++){
		free(rangeIndexes[i]);
		free(rangeSizes[i]);
	}
	for(int i = 0; i < numLayers; i++){
		free(tree[i]);
	}
	free(tree);
	for(int i = 0; i < numPoints; i++){
		free(pointBinNumbers[i]);
	}

	free(pointBinNumbers);
	free(numValidRanges);
	free(calcPerAdd);
	free(addIndexes);
	free(rangeIndexes);
	free(rangeSizes);
	free(A);
	free(binSizes);
	free(binAmounts);
	free(pointArray);
	free(data);
	free(dimensionOrder);
	free(dimOrderedData);
	free(addIndexRange);

    return 1;

}