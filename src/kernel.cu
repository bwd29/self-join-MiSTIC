#include "hip/hip_runtime.h"
#include "include/kernel.cuh"


void launchKernel(int numLayers, double * data, int dim, int numPoints, double epsilon, int * addIndexes, int * addIndexRange, int * pointArray, int ** rangeIndexes, unsigned int ** rangeSizes, int * numValidRanges, unsigned int * numPointsInAdd, unsigned long long *calcPerAdd, int nonEmptyBins, unsigned long long sumCalcs, unsigned long long sumAdds, int * linearRangeIndexes, unsigned int * linearRangeSizes){
 
    double epsilon2 = epsilon*epsilon;
    unsigned long long calcsPerThread = 1000; 

    unsigned int numSearches = pow(3,numLayers);
    unsigned int * numThreadsPerAddress = (unsigned int *)malloc(sizeof(unsigned int)*nonEmptyBins);

    int numBatches = 0;
    unsigned long long threadsPerBatch = KERNEL_BLOCKS * BLOCK_SIZE;
    unsigned long long sum = 0;

    // printf("add 0 calcs: %llu, num Points in that add: %u\n", calcPerAdd[0], numPointsInAdd[0]);
    for(int i = 0; i < nonEmptyBins; i++){
        numThreadsPerAddress[i] = ceil(calcPerAdd[i]*1.0 / calcsPerThread);
        if(numThreadsPerAddress[i] == 0) printf("\nERROR: Threads per address at %d: %u, cals per add: %llu\n",i, numThreadsPerAddress[i], calcPerAdd[i]);
        if (sum + calcPerAdd[i] < calcsPerThread*threadsPerBatch){
            sum += calcPerAdd[i];
        }else{
            sum = calcPerAdd[i];
            numBatches++;
        }
    }

    numBatches++;

    unsigned long long * numCalcsPerBatch = (unsigned long long*)calloc(numBatches,sizeof(unsigned long long));
    unsigned int * numAddPerBatch = (unsigned int*)calloc(numBatches, sizeof(unsigned int));
    unsigned int * numThreadsPerBatch = (unsigned int*)calloc(numBatches,sizeof(unsigned int));
    sum = 0;

    int currentBatch = 0;
    for(int i = 0; i < nonEmptyBins; i++){
        if(numThreadsPerBatch[currentBatch] == 0 || numThreadsPerBatch[currentBatch] + numThreadsPerAddress[i] < threadsPerBatch){
            numThreadsPerBatch[currentBatch] += numThreadsPerAddress[i];
            numAddPerBatch[currentBatch]++;
            numCalcsPerBatch[currentBatch] += calcPerAdd[i];
        } else {
            currentBatch++;
            i = i - 1;
        }

    }

    // for(int i = 0; i < numBatches; i++){
    //     printf("Batch: %d, numThreads: %u, numAdds: %u, numCalcs: %llu\n",i, numThreadsPerBatch[i],numAddPerBatch[i], numCalcsPerBatch[i]);
    // }



    ////////////////////////////////////////////////
    double * d_data;
    assert(hipSuccess == hipMalloc((void**)&d_data, sizeof(double)*numPoints*dim));
    assert(hipSuccess ==  hipMemcpy(d_data, data, sizeof(double)*numPoints*dim, hipMemcpyHostToDevice));

    unsigned int * d_numThreadsPerAddress;
    assert(hipSuccess == hipMalloc((void**)&d_numThreadsPerAddress, sizeof(unsigned int)*nonEmptyBins));
    assert(hipSuccess ==  hipMemcpy(d_numThreadsPerAddress, numThreadsPerAddress, sizeof(unsigned int)*nonEmptyBins, hipMemcpyHostToDevice));

    int * d_addIndexes;
    assert(hipSuccess == hipMalloc((void**)&d_addIndexes, sizeof(int)*nonEmptyBins));
    assert(hipSuccess ==  hipMemcpy(d_addIndexes, addIndexes, sizeof(int)*nonEmptyBins, hipMemcpyHostToDevice));


    int * d_numValidRanges;
    assert(hipSuccess == hipMalloc((void**)&d_numValidRanges, sizeof(int)*nonEmptyBins));
    assert(hipSuccess ==  hipMemcpy(d_numValidRanges, numValidRanges, sizeof(int)*nonEmptyBins, hipMemcpyHostToDevice));

    int * d_rangeIndexes; //double check this for errors
    assert(hipSuccess == hipMalloc((void**)&d_rangeIndexes, sizeof(int)*nonEmptyBins*numSearches));
    assert(hipSuccess ==  hipMemcpy(d_rangeIndexes, linearRangeIndexes, sizeof(int)*numSearches*nonEmptyBins, hipMemcpyHostToDevice));

    unsigned int * d_rangeSizes;
    assert(hipSuccess == hipMalloc((void**)&d_rangeSizes, sizeof(unsigned int)*numSearches*nonEmptyBins));
    assert(hipSuccess ==  hipMemcpy(d_rangeSizes, linearRangeSizes, sizeof(unsigned int)*numSearches*nonEmptyBins, hipMemcpyHostToDevice));


    unsigned int * d_numPointsInAdd;
    assert(hipSuccess == hipMalloc((void**)&d_numPointsInAdd, sizeof(unsigned int)*nonEmptyBins));
    assert(hipSuccess ==  hipMemcpy(d_numPointsInAdd, numPointsInAdd, sizeof(unsigned int)*nonEmptyBins, hipMemcpyHostToDevice));


    int * d_addIndexRange;
    assert(hipSuccess == hipMalloc((void**)&d_addIndexRange, sizeof(int)*nonEmptyBins));
    assert(hipSuccess ==  hipMemcpy(d_addIndexRange, addIndexRange, sizeof(int)*nonEmptyBins, hipMemcpyHostToDevice));


    int * d_pointArray;
    assert(hipSuccess == hipMalloc((void**)&d_pointArray, sizeof(int)*numPoints));
    assert(hipSuccess ==  hipMemcpy(d_pointArray, pointArray, sizeof(int)*numPoints, hipMemcpyHostToDevice));


    unsigned long long int * keyValueIndex;
    assert(hipSuccess == hipHostMalloc((void**)&keyValueIndex, sizeof(unsigned long long int)*numBatches));
    for(int i = 0; i < numBatches; i++){
        keyValueIndex[i] = 0;
    }
    unsigned long long int * d_keyValueIndex;
    assert(hipSuccess == hipMalloc((void**)&d_keyValueIndex, sizeof(unsigned long long int)*numBatches));
    assert(hipSuccess ==  hipMemcpy(d_keyValueIndex, keyValueIndex, sizeof(unsigned long long int)*numBatches, hipMemcpyHostToDevice));


    unsigned int * d_pointA;
    assert(hipSuccess == hipMalloc((void**)&d_pointA, sizeof(unsigned int)*resultsSize));

    unsigned int * d_pointB;
    assert(hipSuccess == hipMalloc((void**)&d_pointB, sizeof(unsigned int)*resultsSize));

    double *d_epsilon2;
    assert(hipSuccess == hipMalloc((void**)&d_epsilon2, sizeof(double)));
    assert(hipSuccess ==  hipMemcpy(d_epsilon2, &epsilon2, sizeof(double), hipMemcpyHostToDevice));

    int *d_dim;
    assert(hipSuccess == hipMalloc((void**)&d_dim, sizeof(int)));
    assert(hipSuccess ==  hipMemcpy(d_dim, &dim, sizeof(int), hipMemcpyHostToDevice));

    unsigned int * d_numThreadsPerBatch;
    assert(hipSuccess == hipMalloc((void**)&d_numThreadsPerBatch, sizeof(unsigned int)*numBatches));
    assert(hipSuccess ==  hipMemcpy(d_numThreadsPerBatch, numThreadsPerBatch, sizeof(unsigned int)*numBatches, hipMemcpyHostToDevice));


    unsigned int * d_numSearches;
    assert(hipSuccess == hipMalloc((void**)&d_numSearches, sizeof(unsigned int)));
    assert(hipSuccess ==  hipMemcpy(d_numSearches, &numSearches, sizeof(unsigned int), hipMemcpyHostToDevice));

    unsigned int * d_numPoints;
    assert(hipSuccess == hipMalloc((void**)&d_numPoints, sizeof(unsigned int)));
    assert(hipSuccess ==  hipMemcpy(d_numPoints, &numPoints, sizeof(unsigned int), hipMemcpyHostToDevice));





    ///////////////////////////////////////////////

    printf("numSearches: %d\n", numSearches);
    
    int batchFirstAdd = 0;
    for(int i = 0; i < numBatches; i++){

        //compute which thread does wich add
        int * addAssign = (int * )malloc(sizeof(int)*numThreadsPerBatch[i]);
        int * threadOffsets = (int*)malloc(sizeof(int)*numThreadsPerBatch[i]);
        unsigned int threadCount = 0;

        for(int j = 0; j < numAddPerBatch[i]; j++){
            if(numThreadsPerAddress[batchFirstAdd + j] == 0) {
                printf("ERROR: add %d has 0 threads\n", j + batchFirstAdd);
                // exit(0);
            }
            for(int k = 0; k < numThreadsPerAddress[batchFirstAdd + j]; k++){
                addAssign[threadCount] = j + batchFirstAdd;
                threadOffsets[threadCount] = k;
                threadCount++;
            }
        }

        batchFirstAdd += numAddPerBatch[i];

        // printf("\nBatch: %d, ThreadCount: %u, ThreadsPerBatch: %u\n",i,threadCount, numThreadsPerBatch[i]);

        /////////////////////////////////////////////////////////

        int * d_addAssign;
        assert(hipSuccess == hipMalloc((void**)&d_addAssign, sizeof(int)*numThreadsPerBatch[i]));
        assert(hipSuccess ==  hipMemcpy(d_addAssign, addAssign, sizeof(int)*numThreadsPerBatch[i], hipMemcpyHostToDevice));


        int * d_threadOffsets;
        assert(hipSuccess == hipMalloc((void**)&d_threadOffsets, sizeof(int)*numThreadsPerBatch[i]));
        assert(hipSuccess ==  hipMemcpy(d_threadOffsets, threadOffsets, sizeof(int)*numThreadsPerBatch[i], hipMemcpyHostToDevice));

        /////////////////////////////////////////////////////////

        hipDeviceSynchronize();

        unsigned int totalBlocks = ceil(numThreadsPerBatch[i]*1.0 / BLOCK_SIZE);


        printf("BatchNumber: %d/%d, Calcs: %llu, Adds: %d, threads: %u, blocks:%d\n ", i+1, numBatches, numCalcsPerBatch[i], numAddPerBatch[i], numThreadsPerBatch[i], totalBlocks);
        
        
        
        
        
        //launch distance kernel
        distanceCalculationsKernel<<<totalBlocks, BLOCK_SIZE>>>(d_numPoints, d_numSearches, d_addAssign, d_threadOffsets, d_epsilon2, d_dim, &d_numThreadsPerBatch[i], d_numThreadsPerAddress, d_data, d_addIndexes, d_numValidRanges, d_rangeIndexes, d_rangeSizes, d_numPointsInAdd, d_addIndexRange, d_pointArray, &d_keyValueIndex[i], d_pointA, d_pointB);

        hipDeviceSynchronize(); 

        assert(hipSuccess ==  hipMemcpy(&keyValueIndex[i], &d_keyValueIndex[i], sizeof(unsigned long long int), hipMemcpyDeviceToHost));

        printf("Results: %llu\n", keyValueIndex[i]);
        //transfer back reuslts

        free(addAssign);
        free(threadOffsets);
        
    }

    unsigned long long totals = 0;
    for(int i = 0; i < numBatches; i++){
        totals += keyValueIndex[i];
    }

    printf("Total results Set Size: %llu\n", totals);

    free(numCalcsPerBatch);
    free(numAddPerBatch);
    free(numThreadsPerBatch);
    free(numThreadsPerAddress);

}

__global__ 
void distanceCalculationsKernel(unsigned int *numPoints, unsigned int *numSearches, int * addAssign, int * threadOffsets, double *epsilon2, int *dim, unsigned int *numThreadsPerBatch, unsigned int * numThreadsPerAddress, double * data, int *addIndexes, int * numValidRanges, int * rangeIndexes, unsigned int * rangeSizes, unsigned int * numPointsInAdd, int * addIndexRange, int * pointArray, unsigned long long int *keyValueIndex, unsigned int * point_a, unsigned int * point_b){

    unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;

    if(tid >= *numThreadsPerBatch){
        return;
    }

    int currentAdd = addAssign[tid]; 
    int threadOffset = threadOffsets[tid];

    for(int i = 0; i < numValidRanges[currentAdd]; i++){
        unsigned long long int numCalcs = rangeSizes[currentAdd*(*numSearches) + i] * numPointsInAdd[currentAdd];
        for(unsigned long long int j = threadOffset; j < numCalcs; j += numThreadsPerAddress[currentAdd]){

            unsigned int pointLocation1 = addIndexRange[currentAdd] + j / rangeSizes[currentAdd*(*numSearches) + i];
            unsigned int pointLocation2 = rangeIndexes[currentAdd*(*numSearches) + i] + j % rangeSizes[currentAdd*(*numSearches) + i];


            if(pointLocation1 > *numPoints) printf("ERROR 1: tid: %d, CurrentAdd: %d, Offset: %d, Point Locations: %u,%u, j: %llu, addVal: %d, size:%u, rangeIndexVal: %d, size:%u\n", tid, currentAdd, threadOffset, pointLocation1,pointLocation2,j,addIndexRange[currentAdd],numPointsInAdd[currentAdd],rangeIndexes[currentAdd*(*numSearches) + i],rangeSizes[currentAdd*(*numSearches) + i]);
            if(pointLocation2 > *numPoints) printf("ERROR 2: tid: %d, CurrentAdd: %d, Offset: %d, Point Locations: %u %u, j: %llu, rangeVal: %d, size: %u\n", tid, currentAdd, threadOffset, pointLocation1, pointLocation2,j,rangeIndexes[currentAdd*(*numSearches) + i],rangeSizes[currentAdd*(*numSearches) + i]);

            unsigned int p1 = pointArray[pointLocation1];
            unsigned int p2 = pointArray[pointLocation2];


            if (distanceCheck((*epsilon2), (*dim), data, p1, p2, (*numPoints))){
                //  store point
                unsigned long long int index = atomicAdd(keyValueIndex,(unsigned long long int)1);
                point_a[index] = p1; //stores the first point Number
                point_b[index] = p2; // this stores the coresponding point number to form a pair
            }
        }
    }
}

__device__ //may need to switch to inline
bool distanceCheck(double epsilon2, int dim, double * data, unsigned int p1, unsigned int p2, unsigned int numPoints){
    double sum = 0;
    for(int i = 0; i < dim; i++){
        #if DATANORM
        sum+=pow(data[i*numPoints + p1] - data[i*numPoints + p2], 2);
        #else
        sum += pow(data[p1*dim+i]-data[p2*dim+i],2);
        #endif
        if(sum > epsilon2) return false;
    }

    return true;
}