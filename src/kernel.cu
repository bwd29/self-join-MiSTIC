#include "hip/hip_runtime.h"
#include "include/kernel.cuh"


void launchKernel(double * data, int dim, int numPoints, double epsilon, int * addIndexes, int * addIndexRange, int * pointArray, int ** rangeIndexes, unsigned int ** rangeSizes, int * numValidRanges, unsigned int * numPointsInAdd, unsigned long long *calcPerAdd, int nonEmptyBins, unsigned long long sumCalcs, unsigned long long sumAdds){
 
    double epsilon2 = epsilon*epsilon;
    unsigned long long calcsPerThread = 100000; //placeholder value of 100k

    unsigned long long * numThreadsPerAddress = (unsigned long long *)malloc(sizeof(unsigned long long)*nonEmptyBins);

    int numBatches = 0;
    unsigned long long threadsPerBatch = KERNEL_BLOCKS * BLOCK_SIZE;
    unsigned long long sum = 0;

    for(int i = 0; i < nonEmptyBins; i++){
        numThreadsPerAddress[i] = ceil(calcPerAdd[i] / calcsPerThread);
        if (sum + calcPerAdd[i] < calcsPerThread*threadsPerBatch || sum == 0){
            sum += calcPerAdd[i];
        }else{
            sum = calcPerAdd[i];
            numBatches++;
        }
    }
    numBatches++;

    unsigned long long * numCalcsPerBatch = (unsigned long long*)malloc(sizeof(unsigned long long)*numBatches);
    unsigned int * numAddPerBatch = (unsigned int*)malloc(sizeof(unsigned int)*numBatches);
    unsigned long long * numThreadsPerBatch = (unsigned long long*)calloc(numBatches,sizeof(unsigned long long));
    sum = 0;
    int batchCount = 0;
    int addCount = 0;
    for(int i = 0; i < nonEmptyBins; i++){
        
        if (sum + calcPerAdd[i] < calcsPerThread*threadsPerBatch || sum == 0){
            numThreadsPerBatch[batchCount] += numThreadsPerAddress[i];
            sum += calcPerAdd[i];
            addCount++;
        }else{
            numCalcsPerBatch[batchCount] = sum;
            numAddPerBatch[batchCount] = addCount;

            sum = calcPerAdd[i];

            addCount = 1;
            batchCount++;

            numThreadsPerBatch[batchCount] += numThreadsPerAddress[i];

        }
    }

    numCalcsPerBatch[numBatches-1] = sum; //for last
    numAddPerBatch[numBatches-1] = addCount;

////////////////////////////////////////////////
double * d_data;
assert(hipSuccess == hipMalloc((void**)&d_data, sizeof(double)*numPoints*dim));
assert(hipSuccess ==  hipMemcpy(d_data, data, sizeof(double)*numPoints*dim, hipMemcpyHostToDevice));

int * d_numThreadsPerAddress;
assert(hipSuccess == hipMalloc((void**)&d_numThreadsPerAddress, sizeof(unsigned long long)*nonEmptyBins));
assert(hipSuccess ==  hipMemcpy(d_numThreadsPerAddress, numThreadsPerAddress, sizeof(unsigned long long)*nonEmptyBins, hipMemcpyHostToDevice));

int * d_addIndexes;
assert(hipSuccess == hipMalloc((void**)&d_addIndexes, sizeof(int)*nonEmptyBins));
assert(hipSuccess ==  hipMemcpy(d_addIndexes, addIndexes, sizeof(int)*nonEmptyBins, hipMemcpyHostToDevice));


int * d_numValidRanges;
assert(hipSuccess == hipMalloc((void**)&d_numValidRanges, sizeof(int)*nonEmptyBins));
assert(hipSuccess ==  hipMemcpy(d_numValidRanges, numValidRanges, sizeof(int)*nonEmptyBins, hipMemcpyHostToDevice));


int ** d_rangeIndexes; //double check this for errors
assert(hipSuccess == hipMalloc((void**)&d_rangeIndexes, sizeof(int*)*nonEmptyBins));
for(int i = 0; i < nonEmptyBins; i++){
    assert(hipSuccess == hipMalloc((void**)&d_rangeIndexes[i], sizeof(int)*numValidRanges[i]));
    assert(hipSuccess ==  hipMemcpy(d_rangeIndexes[i], rangeIndexes[i], sizeof(int)*numValidRanges[i], hipMemcpyHostToDevice));
}

unsigned int ** d_rangeSizes;
assert(hipSuccess == hipMalloc((void**)&d_rangeSizes, sizeof(unsigned int*)*nonEmptyBins));
for(int i = 0; i < nonEmptyBins; i++){
    assert(hipSuccess == hipMalloc((void**)&d_rangeSizes[i], sizeof(unsigned int)*numValidRanges[i]));
    assert(hipSuccess ==  hipMemcpy(d_rangeSizes[i], rangeSizes[i], sizeof(unsigned int)*numValidRanges[i], hipMemcpyHostToDevice));

}

unsigned int * d_numPointsInAdd;
assert(hipSuccess == hipMalloc((void**)&d_numPointsInAdd, sizeof(unsigned int)*nonEmptyBins));
assert(hipSuccess ==  hipMemcpy(d_numPointsInAdd, numPointsInAdd, sizeof(unsigned int)*nonEmptyBins, hipMemcpyHostToDevice));


int * d_addIndexRange;
assert(hipSuccess == hipMalloc((void**)&d_addIndexRange, sizeof(int)*nonEmptyBins));
assert(hipSuccess ==  hipMemcpy(d_addIndexRange, addIndexRange, sizeof(int)*nonEmptyBins, hipMemcpyHostToDevice));


int * d_pointArray;
assert(hipSuccess == hipMalloc((void**)&d_pointArray, sizeof(int)*numPoints));
assert(hipSuccess ==  hipMemcpy(d_pointArray, pointArray, sizeof(int)*numPoints, hipMemcpyHostToDevice));


unsigned long long * keyValueIndex = (unsigned long long *)calloc(numBatches, sizeof(unsigned long long ));
unsigned long long * d_keyValueIndex;
assert(hipSuccess == hipMalloc((void**)&d_keyValueIndex, sizeof(unsigned long long)*numBatches));
assert(hipSuccess ==  hipMemcpy(d_keyValueIndex, keyValueIndex, sizeof(unsigned long long)*numBatches, hipMemcpyHostToDevice));


unsigned int * pointA;
assert(hipSuccess == hipMalloc((void**)&pointA, sizeof(unsigned int)*resultsSize));

unsigned int * pointB;
assert(hipSuccess == hipMalloc((void**)&pointB, sizeof(unsigned int)*resultsSize));



///////////////////////////////////////////////
    

    for(int i = 0; i < numBatches; i++){

        const double d_epsilon2 = epsilon2;
        const int d_dim = dim;
        const int d_numThreadsPerBatch = numThreadsPerBatch[i];

        //compute which thread does wich add
        int * addAssign = (int * )malloc(sizeof(int)*numThreadsPerBatch[i]);
        int * threadOffsets = (int*)malloc(sizeof(int)*numThreadsPerBatch[i]);
        unsigned int currentAdd = 0;
        unsigned int offsetCount = 0;

        for(unsigned int j = 0; j < numThreadsPerBatch[i]; j++){
            // if(currentAdd > nonEmptyBins) printf("current add is to large!");
            if ( offsetCount > numThreadsPerAddress[currentAdd]){
                currentAdd++;
                offsetCount = 0;
            }
            addAssign[j] = currentAdd;
            threadOffsets[j] = offsetCount;
            offsetCount++;
        }

        /////////////////////////////////////////////////////////

        int * d_addAssign;
        assert(hipSuccess == hipMalloc((void**)&d_addAssign, sizeof(int)*numThreadsPerBatch[i]));

        int * d_threadOffsets;
        assert(hipSuccess == hipMalloc((void**)&d_threadOffsets, sizeof(int)*numThreadsPerBatch[i]));

        /////////////////////////////////////////////////////////

        unsigned int totalBlocks = ceil(numThreadsPerBatch[i] / BLOCK_SIZE);


        printf("BatchNumber: %d/%d, Calcs: %llu, Adds: %d, threads: %llu, blocks:%d\n", i+1, numBatches, numCalcsPerBatch[i], numAddPerBatch[i], numThreadsPerBatch[i], totalBlocks);
        
        
        
        
        
        //launch distance kernel
        //distanceCalculationsKernel<<<KERNEL_BLOCKS, BLOCK_SIZE>>>(int * addAssign, int * threadOffsets, const double epsilon2, const int dim, const int numThreadsPerBatch, int * numThreadsPerAddress, double * data, int *addIndexes, int * numValidRanges, int ** rangeIndexes, unsigned int ** rangeSizes, unsigned int * numPointsInAdd, int * addIndexRange, int * pointArray, unsigned long long *keyValueIndex, unsigned int * point_a, unsigned int * point_b);


        //transfer back reuslts

        free(addAssign);
        free(threadOffsets);
        
    }

    free(numCalcsPerBatch);
    free(numAddPerBatch);
    free(numThreadsPerBatch);
    free(numThreadsPerAddress);

}

__device__ 
void distanceCalculationsKernel(int * addAssign, int * threadOffsets, const double epsilon2, const int dim, const int numThreadsPerBatch, int * numThreadsPerAddress, double * data, int *addIndexes, int * numValidRanges, int ** rangeIndexes, unsigned int ** rangeSizes, unsigned int * numPointsInAdd, int * addIndexRange, int * pointArray, unsigned long long *keyValueIndex, unsigned int * point_a, unsigned int * point_b){

    unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;

    if(tid > numThreadsPerBatch){
        return;
    }

    int currentAdd = addAssign[tid];
    int threadOffset = threadOffsets[tid];

    for(int i = 0; i < numValidRanges[currentAdd]; i++){
        for(int j = 0; j < rangeSizes[currentAdd][i] * numPointsInAdd[currentAdd] + threadOffset; j += numThreadsPerAddress[currentAdd]){
            unsigned int p1 = pointArray[addIndexRange[currentAdd] + j/rangeSizes[currentAdd][i]];
            unsigned int p2 = pointArray[rangeIndexes[currentAdd][i] + j % rangeSizes[currentAdd][i]];
            if (distanceCheck(epsilon2, dim, &data[p1], &data[p2])){
                //store point
                unsigned int index = atomicAdd(keyValueIndex,(unsigned int)1);
                point_a[index] = p1; //stores the first point Number
                point_b[index] = p2; // this stores the cooresponding point number to form a pair
            }
        }
    }
}

__device__ //may need to switch to inline
bool distanceCheck(double epsilon2, double dim, double * p1, double * p2){
    double sum = 0;
    for(int i = 0; i < dim; i++){
        sum += pow(p1[i]-p2[i],2);
        if(sum >= epsilon2) return false;
    }

    return true;
}