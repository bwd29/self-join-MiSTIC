#include "hip/hip_runtime.h"
#include "include/utils.cuh"


int * stddev( double * A, int dim, int num_points) {
	double mean, devmean;
	double *deviation = (double*)malloc(sizeof(double) * dim);
	int *dimension = (int*)malloc(sizeof(int) * dim);
	for(int i = 0; i < dim; i++) {
		dimension[i] = i;
	}
	for(int i = 0; i < dim; i++){
		mean = 0.0;
		for(int j = 0; j < num_points; j++){
			mean += A[dim*j+i];
		}
		mean /= num_points;
		devmean = 0.0;
		for(int j = 0; j < num_points; j++){
			devmean += pow(A[dim*j + i] - mean,2);
		}
		devmean /= num_points;
		deviation[i] = sqrt(devmean);
	}
	thrust::sort_by_key(deviation, &deviation[dim-1], dimension);
	double *deviationret = (double*)malloc(sizeof(double) * dim);
	int *dimensionret = (int*)malloc(sizeof(int) * dim);
	for(int i = 0; i < dim; i++){
		deviationret[i] = deviation[dim-1-i];
		dimensionret[i] = dimension[dim-1-i];
	}
	free(deviationret);
	free(deviation);
	free(dimension);
	return dimensionret;
}

double euclideanDistance(double * dataPoint, int dim, double * RP){
    // get the euclidean distance
    double distance = 0;
    for(int i = 0; i < dim; i++){
        double diff = (RP[i] - dataPoint[i]);
        distance += diff * diff;
    }
    distance = sqrt(distance);
    return distance;
}

double * createRPArray(double * data, int numRP, int dim, unsigned long long numPoints){

	int sample_size = numPoints*SAMPLE_PER;

	double * testRPArray = new double[TEST_RP*dim];

	//randomly place the rps
	// #pragma omp parallel for
	for(int i = 0; i < TEST_RP*dim; i++){
		testRPArray[i] = (double)rand()/(double)RAND_MAX;
	}

	//get the distances
	double *distmat = new double[TEST_RP*sample_size];

	// #pragma omp parallel for
	for(int i = 0; i < sample_size; i++){
		for(int j = 0; j < TEST_RP; j++){
			distmat[i*TEST_RP+j] = euclideanDistance(&data[i*dim], dim, &testRPArray[j*dim]);
		}
	}

	//get std dev of dist mat
	int * order = stddev(distmat, TEST_RP, sample_size);

	//get first numRP rps
	double * RPArray = (double *)malloc(sizeof(double)*numRP*dim);

	// #pragma omp parallel for
	for(int i = 0; i < numRP; i++){
		for(int j = 0; j < dim; j++){
			RPArray[i*dim+j] = testRPArray[ order[i]*dim + j ];
		}
	}

	// delete(testRPArray);
	// delete(distmat);

    return RPArray;
}
