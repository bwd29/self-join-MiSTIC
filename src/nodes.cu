#include "hip/hip_runtime.h"

#include "include/nodes.cuh"

//function to build up net of nodes
unsigned int buildNodeNet(double * data,
                 unsigned int dim,
                 unsigned int numPoints,
                 unsigned int numRP,
                 unsigned int * pointArray,
                 double epsilon,
                 std::vector<struct Node> * outNodes){


    hipSetDevice(CUDA_DEVICE);

    std::vector<std::vector<struct Node>> subGraph;
    std::vector<struct Node> newNodes;
    unsigned int numNodes;
    double calcTime;
    double nodePerSecond;
    unsigned int numSplits = 0;
    double previousCalcTime = 0;
    double predictedNodeTime = 0;
    unsigned long long int  calcsPerSecond;


    struct DevicePointers devicePointers;

#if DEVICE_BUILD
    double * d_data;
    assert(hipSuccess == hipMalloc((void**)&d_data, sizeof(double)*numPoints*dim));
    assert(hipSuccess ==  hipMemcpy(d_data, data, sizeof(double)*numPoints*dim, hipMemcpyHostToDevice));

    unsigned int *d_dim;
    assert(hipSuccess == hipMalloc((void**)&d_dim, sizeof(unsigned int)));
    assert(hipSuccess ==  hipMemcpy(d_dim, &dim, sizeof(unsigned int), hipMemcpyHostToDevice));

    // copy over the number of points in the dataset
    unsigned int * d_numPoints;
    assert(hipSuccess == hipMalloc((void**)&d_numPoints, sizeof(unsigned int)));
    assert(hipSuccess ==  hipMemcpy(d_numPoints, &numPoints, sizeof(unsigned int), hipMemcpyHostToDevice));

    // copy over the number of points in the dataset
    double * d_epsilon;
    assert(hipSuccess == hipMalloc((void**)&d_epsilon, sizeof(double)));
    assert(hipSuccess ==  hipMemcpy(d_epsilon, &epsilon, sizeof(double), hipMemcpyHostToDevice));

    
    devicePointers.d_data = d_data;
    devicePointers.d_dim = d_dim;
    devicePointers.d_numPoints = d_numPoints;
    devicePointers.d_epsilon = d_epsilon;

#endif

    unsigned long long int previousDistCalcs = numPoints*numPoints;
    unsigned int numPreviousNodes = 0;

    // need to go through each reference point
    for(unsigned int i = 0; i < MAXRP; i++){ 

        // generate some reference points
        double * RPArray = createRPArray(data, RPPERLAYER, dim, numPoints);
    
        std::vector<std::vector<struct Node>> layerNodes;
        layerNodes.resize(RPPERLAYER);
        unsigned long long int lowestDistCalcs = ULLONG_MAX;
        unsigned int bestRP = 0;

        
        #if DEVICE_BUILD

        unsigned int * allBinNumber = (unsigned int * )malloc(sizeof(unsigned int)*numPoints*RPPERLAYER);

        //create bin number arrays on device
        unsigned int  * d_binNumber;
        assert(hipSuccess == hipMalloc((void**)&d_binNumber, sizeof(unsigned int)*numPoints*RPPERLAYER));

        double * d_RP;
        assert(hipSuccess == hipMalloc((void**)&d_RP, sizeof(double)*dim*RPPERLAYER));
        assert(hipSuccess == hipMemcpy(d_RP, RPArray, sizeof(double)*dim*RPPERLAYER, hipMemcpyHostToDevice));

        hipStream_t stream;
        hipError_t stream_check = hipStreamCreate(&stream);
        assert(hipSuccess == stream_check);

        unsigned int totalBlocks = ceil(numPoints*1.0/1024);

        double cT1 = omp_get_wtime();
        binningKernel<<<totalBlocks, 1024, 0, stream>>>(d_binNumber,
                                                                devicePointers.d_numPoints,
                                                                devicePointers.d_dim,
                                                                devicePointers.d_data,
                                                                d_RP,
                                                                devicePointers.d_epsilon);

        hipStreamSynchronize(stream);

        double cT2 = omp_get_wtime();

        if(i==0){
            calcsPerSecond = (unsigned long long int) numPoints*RPPERLAYER / (cT2-cT1) * sqrt(dim);//CALC_MULTI;
        } else {
            calcsPerSecond += (unsigned long long int) numPoints*RPPERLAYER / (cT2-cT1) * sqrt(dim);//CALC_MULTI;
            calcsPerSecond = calcsPerSecond / 2;
        }
        printf("Predicted calcsPerSecond: %llu\n", calcsPerSecond);
        

        assert(hipSuccess == hipMemcpyAsync(allBinNumber, d_binNumber, sizeof(unsigned int)*numPoints*RPPERLAYER, hipMemcpyDeviceToHost, stream));

        hipStreamSynchronize(stream);

        #endif  

        std::vector<std::vector<struct Node>> tempGraph;

        // itterate through all of the subgraphs
        unsigned int numSubs;
        if(subGraph.size() == 0){
            numSubs = 1;
        } else {
            numSubs = subGraph.size();
        }

        lowestDistCalcs = 0;
        // printf("Num subs to gen: %u\n", numSubs);
        for(unsigned int n = 0; n < numSubs ; n++){
            unsigned long long int subLowestDistCalcs = ULLONG_MAX;


            #pragma omp parallel for num_threads(RPPERLAYER)
            for(unsigned int j = 0; j < RPPERLAYER; j++){
                // need to compare num dist calcs for different potental RP
                std::vector<struct Node> tempNodes;// subGraph[n];
                unsigned int tempNumNodes = 0;
                double tempCalcTime;
                double tempNodePerSecond;
                
                if(i == 0){
                    unsigned int * tempPointArray = (unsigned int *)malloc(sizeof(unsigned int)*numPoints);
                    #pragma omp parallel for num_threads(8)
                    for(unsigned int k = 0; k < numPoints; k++){
                        tempPointArray[k] = pointArray[k];
                    } 
                    tempNumNodes = initNodes(data, dim, numPoints, epsilon, &allBinNumber[numPoints*j], tempPointArray, &layerNodes[j], devicePointers, &tempCalcTime);
                    free(tempPointArray);
                }
                else{
                    tempNodes = subGraph[n];
                    tempNumNodes = splitNodes(&allBinNumber[numPoints*j], tempNodes, tempNodes.size(), epsilon, data, dim, numPoints, &layerNodes[j], devicePointers, &tempNodePerSecond);
                    // printf("    subgraph %u with %u origional nodes; RP %u has %u nodes\n", n, tempNodes.size(), j, tempNumNodes );
                }
                
                // printf("check: %llu\n", tempNodes[0].numCalcs);
                
                unsigned long long numCalcs = totalNodeCalcs(layerNodes[j], tempNumNodes);
                // unsigned long long sumSqrs = nodeSumSqrs(layerNodes[j], tempNumNodes);
                // printf("    Layer %d for RP %d has Nodes: %u with calcs: %llu\n", i, j, tempNumNodes, numCalcs);

                #pragma omp critical
                {
                    if(numCalcs < subLowestDistCalcs){
                        subLowestDistCalcs = numCalcs;
                        bestRP = j;
                        // layerNodes = tempNodes;
                        numNodes = tempNumNodes;
                        if(i==0){
                            calcTime = tempCalcTime;
                            nodePerSecond = 2000;
                        } 
                        if(i!=0) nodePerSecond = tempNodePerSecond;
                    }
                }
                
            }


            printf("SubGraph %u Layer %d Selecting RP %d with Nodes: %u and calcs: %llu :: ", n, i, bestRP, numNodes, subLowestDistCalcs);
            
            #if SUBG
            std::vector<std::vector<struct Node>> layerSubGraphs = genSubGraphs(layerNodes[bestRP]);
            tempGraph.insert(tempGraph.end(), layerSubGraphs.begin(), layerSubGraphs.end());
            #else
            tempGraph.push_back(layerNodes[bestRP]);
            #endif
       
            lowestDistCalcs += subLowestDistCalcs;
        }

        numNodes = 0; 
        for(unsigned int j = 0; j < tempGraph.size(); j++){
            numNodes += tempGraph[j].size();
        }

        subGraph = tempGraph;


        // printf("Layer %d Selecting RP %d with Nodes: %u and calcs: %llu\n", i, bestRP, numNodes, lowestDistCalcs);

        
    
        double actualNodeTime = numNodes*1.0 / nodePerSecond + (cT2-cT1);

        printf("Predicted time: %f, Actual Time: %f\n", predictedNodeTime, actualNodeTime);

        // newNodes = layerNodes[bestRP];
        
        // subGraph = genSubGraphs(newNodes);

        //lowestDistCalcs*LAYER_DIFF > previousDistCalcs 

        double calcRatio = lowestDistCalcs*1.0/ previousDistCalcs;
        double nodeRatio = numNodes/ numPreviousNodes;
        calcTime = lowestDistCalcs*1.0 / calcsPerSecond;
        double timeReduction = previousCalcTime - calcTime;
        predictedNodeTime = numNodes*1.0 / nodePerSecond + (cT2-cT1);

        #if DEVICE_BUILD
        // double calcsPerSecondDyn = calcsPerSecond;//numPoints / calcTime;
        printf("Build Time: %f, Calc Time: %f, reduction %f\n############################################\n", predictedNodeTime, calcTime, timeReduction);
        // if(i > MINRP && ( newNodes.size()*1.0 / nodePerSecond *10> lowestDistCalcs*1.0 / calcsPerSecondDyn || i >= MAXRP)){ 
        if(i >= MINRP && ( predictedNodeTime > timeReduction || i >= MAXRP)){ 

            // printf("\nPrevious Calcs: %llu, Current: %llu, ratio: %f\n", previousDistCalcs, lowestDistCalcs, calcRatio);
            // printf("Previous Nodes: %u, Current Nodes: %u, ratio: %f\n",numPreviousNodes, newNodes.size(), nodeRatio);
            numSplits = i+1;
            break;
        }else{
            numSplits = i+1;
            previousCalcTime = calcTime;
            previousDistCalcs = lowestDistCalcs;
            numPreviousNodes = numNodes;
        }
        #else
        printf("Build Time: %f, Calc Time: %f, reduction %f\n", predictedNodeTime, calcTime, timeReduction);
        // if(i > MINRP && ( newNodes.size()*1.0 / nodePerSecond *10> lowestDistCalcs*1.0 / calcsPerSecond || i >= MAXRP)){ 
        if(i > MINRP && ( predictedNodeTime > timeReduction || i >= MAXRP)){ 

            // printf("\nPrevious Calcs: %llu, Current: %llu, ratio: %f\n", previousDistCalcs, lowestDistCalcs, calcRatio);
            // printf("Previous Nodes: %u, Current Nodes: %u, ratio: %f\n",numPreviousNodes, newNodes.size(), nodeRatio);
            numSplits = i+1;
            break;
        }else{
            previousCalcTime = calcTime;
            previousDistCalcs = lowestDistCalcs;
            numPreviousNodes = numNodes;
            numSplits = i+1;
        }
        #endif
    }




    //linearize the sub graphs

    #if SUBG
    unsigned int nodeCounter = 0;
    for(unsigned int i = 0; i < subGraph.size(); i++){
        for(unsigned int j = 0; j < subGraph[i].size(); j++){
            newNodes.push_back(subGraph[i][j]);
            newNodes[nodeCounter].nodeIndex = nodeCounter;
            subGraph[i][j].nodeIndex = nodeCounter;
            nodeCounter++;
        }
    }

    nodeCounter = 0;
    unsigned int largestSub = subGraph[0].size();
    for(unsigned int i = 0; i < subGraph.size(); i++){
        if(subGraph[i].size() > largestSub) largestSub = subGraph[i].size();
        for(unsigned int j = 0; j < subGraph[i].size(); j++){
            for(unsigned int k = 0; k < subGraph[i][j].neighborIndex.size(); k++){
                newNodes[nodeCounter].neighborIndex[k] = subGraph[i][subGraph[i][j].neighborIndex[k]].nodeIndex;
            }
            nodeCounter++;
        }
    }

        // printf("check: %llu\n", newNodes[0].numCalcs);
        unsigned long long numCalcs = totalNodeCalcs(newNodes, newNodes.size());
        // unsigned long long sumSqrs = nodeSumSqrs(newNodes, newNodes.size());
    
        printf("Final graph has %u nodes, %u subgraphs, largest sub: %u, %llu calcs total\n", numNodes,subGraph.size(), largestSub, numCalcs);
    
        if(ERRORPRINT) fprintf(stderr,"%u %u %u %u %llu ", subGraph.size(), largestSub, numSplits, numNodes, numCalcs);

    #else
        newNodes = subGraph[0];

        unsigned long long numCalcs = totalNodeCalcs(newNodes, newNodes.size());
        // unsigned long long sumSqrs = nodeSumSqrs(newNodes, newNodes.size());

        printf("Final graph has %u nodes, %llu calcs total\n", numNodes, numCalcs);

        if(ERRORPRINT) fprintf(stderr,"%u %u %llu ", numSplits, numNodes, numCalcs);
    #endif

    // printf("check: %llu\n", newNodes[0].numCalcs);



    //rearange the pointArray
    unsigned int counter = 0;
    for(unsigned int i = 0; i < newNodes.size(); i++){
        //append own index to neighbors
        newNodes[i].pointOffset = counter;
        for(unsigned int j = 0; j < newNodes[i].numNodePoints; j++){
            pointArray[counter] = newNodes[i].nodePoints[j];
            counter++;
        }
    }

    *outNodes = newNodes;

#if DEVICE_BUILD
    hipFree(d_data);
    hipFree(d_dim);
    hipFree(d_numPoints);
    hipFree(d_epsilon);
#endif

    return numNodes;

}

unsigned int initNodes(double * data,
                        unsigned int dim,
                        unsigned int numPoints,
                        double epsilon,
                        unsigned int * binNumber,
                        unsigned int * pointArray,
                        std::vector<struct Node> * nodes,
                        struct DevicePointers devicePointers,
                        double * calcTime){


    hipSetDevice(CUDA_DEVICE);
    std::vector<struct Node> newNodes;

#if DEVICE_BUILD

//     unsigned int * d_pointArray;
//     assert(hipSuccess == hipMalloc((void**)&d_pointArray, sizeof(unsigned int)*numPoints));
//     assert(hipSuccess == hipMemcpy(d_pointArray, pointArray, sizeof(unsigned int)*numPoints, hipMemcpyHostToDevice));

//     //create bin number arrays on device
//     unsigned int  * d_binNumber;
//     assert(hipSuccess == hipMalloc((void**)&d_binNumber, sizeof(unsigned int)*numPoints));
    
//     double * d_RP;
//     assert(hipSuccess == hipMalloc((void**)&d_RP, sizeof(double)*dim));
//     assert(hipSuccess == hipMemcpy(d_RP, RP, sizeof(double)*dim, hipMemcpyHostToDevice));

//     hipStream_t stream;
//     hipError_t stream_check = hipStreamCreate(&stream);
//     assert(hipSuccess == stream_check);

//     unsigned int totalBlocks = ceil(numPoints*1.0/BLOCK_SIZE);

//     double time1 = omp_get_wtime();
    
//     binningKernel<<<totalBlocks, BLOCK_SIZE, 0, stream>>>(d_binNumber,
//                                                             devicePointers.d_numPoints,
//                                                             devicePointers.d_dim,
//                                                             devicePointers.d_data,
//                                                             d_RP,
//                                                             devicePointers.d_epsilon);

//     hipStreamSynchronize(stream);

//     double time2 = omp_get_wtime();

//     *calcTime = time2-time1;
    // sort the node points based on their bin numbers
    thrust::sort_by_key(thrust::omp::par, &binNumber[0], &binNumber[numPoints], &pointArray[0]);

    // thrust::sort_by_key(thrust::cuda::par.on(stream), d_binNumber, d_binNumber + numPoints, pointArray);

    // hipStreamSynchronize(stream);

    // assert(hipSuccess == hipMemcpyAsync(binNumber, d_binNumber, sizeof(unsigned int)*numPoints, hipMemcpyDeviceToHost, stream));

    // assert(hipSuccess == hipMemcpyAsync(pointArray, d_pointArray, sizeof(unsigned int)*numPoints, hipMemcpyDeviceToHost, stream));

    // hipStreamSynchronize(stream);

#else


    // // #pragma omp parallel for
    for(unsigned int i = 0; i < numPoints; i++){
        //get distance of each point in the node to the reference point
        binNumber[i] = floor( euclideanDistance(&data[i*dim],dim,RP) / epsilon);
    }

        thrust::sort_by_key(thrust::host, &binNumber[0], &binNumber[numPoints], &pointArray[0]);


#endif


    //if all the points are in the same bin
    if(binNumber[0] == binNumber[numPoints-1]){

        newNodes.push_back(newNode(numPoints, pointArray, binNumber[0], 0));

        updateNodeCalcs(&newNodes, newNodes.size());
        //free temp memory
        // free(binNumber);

        *nodes = newNodes;
        //go to the next node
        return 1;
    }

    //go through and make nodes

    //variable to keep track of last bin end
    unsigned int tempBinPointer = 0;

    //variable to count new nodes
    unsigned int numNewNodes = 0;

    unsigned int bcounter = 0;
    //scan through and create a new node for each non-empty bin
    for(unsigned int i = 0; i < numPoints; i++){
        bcounter++;
        //check if need to make a new node
        if(i == numPoints-1 || binNumber[i] != binNumber[i+1]){
            // printf("making new node, j: %d, tempBinPointer: %d, numPoints in the new node:%d\n", i, tempBinPointer, i - tempBinPointer+1 );
            // if(i== numPoints - 1) {
            //     printf("BinNumber#%u: %u->%u: p=%u->%u\n", numNewNodes, binNumber[i], 0 ,bcounter,i-tempBinPointer+1);
            // }else{ 
            //     printf("BinNumber#%u: %u->%u: p=%u->%u\n", numNewNodes, binNumber[i], binNumber[i+1], bcounter,i-tempBinPointer+1);
            // }
            //push back the new node onto the temporary vector of nodes
            newNodes.push_back( newNode(i-tempBinPointer+1, pointArray+tempBinPointer, binNumber[i], numNewNodes ) );
            tempBinPointer = i+1;
            numNewNodes++;
            bcounter = 0;
        }
    }

    //create the connections
            //now that the split nodes exist, modify neighbor values
    //special case for the first
    if(newNodes[0].binNumbers.back() == newNodes[1].binNumbers.back() - 1 ){ //already know at least 2 nodes in list
        newNodes[0].neighborIndex.push_back(1);
    } 

    // handle all middle nodes
    for(unsigned int i = 1; i < numNewNodes-1; i++){
        //check if lower bin is one away
        if(newNodes[i].binNumbers.back() == newNodes[i-1].binNumbers.back() + 1){
            newNodes[i].neighborIndex.push_back(i-1);
        }
        //check if upper bin is one away
        if(newNodes[i].binNumbers.back() == newNodes[i+1].binNumbers.back() - 1){
            newNodes[i].neighborIndex.push_back(i+1);
        }
    }

    //special case for last node
    if(newNodes[numNewNodes-1].binNumbers.back() == newNodes[numNewNodes-2].binNumbers.back() + 1 ){ //already know at least 2 nodes in list
        newNodes[numNewNodes-1].neighborIndex.push_back(numNewNodes-2);
    }


    updateNodeCalcs(&newNodes, newNodes.size());
    //assign the vector to the return
 
    // printf("check: %llu\n", newNodes[0].numCalcs);

    *nodes = newNodes;
    
    // free(binNumber);
    
#if DEVICE_BUILD
    // hipFree(d_binNumber);
    // hipFree(d_RP);
    // hipFree(d_pointArray);
#endif
  
    return numNewNodes;

}

//splits a node based on a reference point and return the number of new nodes
unsigned int splitNodes(unsigned int * allBinNumbers, //the reference point used for the split
                    std::vector<struct Node> nodes,// the array of nodes
                    unsigned int numNodes,//the number of nodes
                    double epsilon, //the distance threshold of the search
                    double * data, //the dataset
                    unsigned int dim,//the number of dimensions of the data
                    unsigned int numPoints,// number of points in the dataset
                    std::vector<struct Node> * newNodes,
                    struct DevicePointers devicePointers,
                    double * nodePerSecond){  // pointer for returning the new nodes
    
    hipSetDevice(CUDA_DEVICE);
    double time1 = omp_get_wtime();

    // printf("Start split\n");
    //need to keep track of all of the new split nodes
    std::vector<std::vector<struct Node>> tempNewNodes;
    tempNewNodes.resize(numNodes);

// #if DEVICE_BUILD
    
//     unsigned int * allBinNumber = (unsigned int * )malloc(sizeof(unsigned int)*numPoints);

//     //create bin number arrays on device
//     unsigned int  * d_binNumber;
//     assert(hipSuccess == hipMalloc((void**)&d_binNumber, sizeof(unsigned int)*numPoints));
    
//     double * d_RP;
//     assert(hipSuccess == hipMalloc((void**)&d_RP, sizeof(double)*dim));
//     assert(hipSuccess == hipMemcpy(d_RP, RP, sizeof(double)*dim, hipMemcpyHostToDevice));

//     hipStream_t stream;
//     hipError_t stream_check = hipStreamCreate(&stream);
//     assert(hipSuccess == stream_check);

//     unsigned int totalBlocks = ceil(numPoints*1.0/BLOCK_SIZE);

//     binningKernel<<<totalBlocks, BLOCK_SIZE, 0, stream>>>(d_binNumber,
//                                                             devicePointers.d_numPoints,
//                                                             devicePointers.d_dim,
//                                                             devicePointers.d_data,
//                                                             d_RP,
//                                                             devicePointers.d_epsilon);

//     hipStreamSynchronize(stream);

//     assert(hipSuccess == hipMemcpyAsync(allBinNumber, d_binNumber, sizeof(unsigned int)*numPoints, hipMemcpyDeviceToHost, stream));

//     hipStreamSynchronize(stream);

// #endif

    // printf("allocated vec for %d nodes\n", numNodes);
    // go through each node and split
    for(unsigned int i = 0; i < numNodes; i++){

        if(nodes[i].numCalcs < MAX_CALCS_PER_NODE*1000000/*MIN_NODE_SIZE */ ){
            std::vector<struct Node> tempNodes;
            tempNodes.push_back(newNode(nodes[i].numNodePoints, &(nodes[i].nodePoints[0]), nodes[i], (unsigned int) -1, 0 ) );
            tempNodes[0].split = false;
            tempNewNodes[i] = tempNodes; 
            continue;
        }

        // printf("For node %d, starting binning\n", i);
        //temp array to hold each points new bin number
        unsigned int * binNumber = (unsigned int * )malloc(sizeof(unsigned int)*nodes[i].numNodePoints);


        // break nodes into new nodes
        // #pragma omp parallel for
        for(unsigned int j = 0; j < nodes[i].numNodePoints; j++){
            //get distance of each point in the node to the reference point
            #if DEVICE_BUILD
            binNumber[j] = allBinNumbers[nodes[i].nodePoints[j]];
            #else
            binNumber[j] = floor( euclideanDistance(&data[nodes[i].nodePoints[j]*dim],dim,RP) / epsilon);
            #endif
        }

        // printf("finished binning\n");


        // sort the node points based on their bin numbers
        thrust::sort_by_key(thrust::omp::par, &binNumber[0], &binNumber[nodes[i].numNodePoints], &nodes[i].nodePoints[0]);
        
        // printf("finished sorting\n");
        
        //temp vector to hold new nodes
        std::vector<struct Node> tempNodes;


        //if all the points are in the same bin
        if(binNumber[0] == binNumber[nodes[i].numNodePoints-1]){
            // printf("no splits\n");
            //add the bin number
            // nodes[i].binNumbers.push_back(binNumber[0]);
            tempNodes.push_back(newNode(nodes[i].numNodePoints, &(nodes[i].nodePoints[0]), nodes[i], binNumber[0], 0 ) );
            tempNewNodes[i] = tempNodes;
            //free temp memory
            free(binNumber);

            //go to the next node
            continue;
        }

        // printf("finished same bin Check\n");




        //variable to keep track of last bin end
        unsigned int tempBinPointer = 0;

        //variable to count new nodes
        unsigned int numNewNodes = 0;

        unsigned int bcounter = 0;

        //scan through and create a new node for each non-empty bin
        // printf("NumNodePoints for node %u: %u\n", i,  nodes[i].numNodePoints);
        for(unsigned int j = 0; j < nodes[i].numNodePoints; j++){
            bcounter++;
            
            //check if need to make a new node
            if(j == nodes[i].numNodePoints-1 || binNumber[j] != binNumber[j+1]){
                
                // printf("making new node, j: %d, numNodePoints: %d, tempBinPointer: %d, numPoints in the new node:%d\n", j,nodes[i].numNodePoints, tempBinPointer, j - tempBinPointer+1 );
                //push back the new node onto the temporary vector of nodes
                // if(j== nodes[i].numNodePoints - 1) {
                //     printf("BinNumber#%u: %u->%u: p=%u->%u; j: %u\n", numNewNodes, binNumber[j], 0 ,bcounter,j-tempBinPointer+1, j);
                // }else{ 
                //     printf("BinNumber#%u: %u->%u: p=%u->%u; j: %u\n", numNewNodes, binNumber[j], binNumber[j+1], bcounter,j-tempBinPointer+1, j);
                // }
                tempNodes.push_back( newNode(j-tempBinPointer+1, &(nodes[i].nodePoints[0]) + tempBinPointer, nodes[i], binNumber[j], numNewNodes ) );
                tempBinPointer = j+1;
                numNewNodes++;
                bcounter = 0;
            }
        }

        // printf("finished creating new nodes for non empty bins: new nodes : %u\n", numNewNodes);

        //now that the split nodes exist, modify neighbor values
        //special case for the first
        if(tempNodes[0].binNumbers.back() == tempNodes[1].binNumbers.back() - 1 ){ //already know at least 2 nodes in list
            tempNodes[0].neighborIndex.push_back(1);
        }

        // handle all middle nodes
        for(unsigned int j = 1; j < numNewNodes-1; j++){
            //check if lower bin is one away
            if(tempNodes[j].binNumbers.back() == tempNodes[j-1].binNumbers.back() + 1){
                tempNodes[j].neighborIndex.push_back(j-1);
            }
            //check if upper bin is one away
            if(tempNodes[j].binNumbers.back() == tempNodes[j+1].binNumbers.back() - 1){
                tempNodes[j].neighborIndex.push_back(j+1);
            }
        }

        //special case for last node
        if(tempNodes[numNewNodes-1].binNumbers.back() == tempNodes[numNewNodes-2].binNumbers.back() + 1 ){ //already know at least 2 nodes in list
            tempNodes[numNewNodes-1].neighborIndex.push_back(numNewNodes-2);
        }

        
        //copy the temp nodes back into the larger vector of nodes
        tempNewNodes[i]  = tempNodes;
        free(binNumber);

    }
    // printf("finished intital splitting\n");

    ////////////////////////////////////////////////////////////////
    // All nodes in the temp nodes now point to their neighbors within the smaller array
    // the neighbor index values will have to be updated when merging with the other vectors
    /////////////////////////////////////////////////////////////

    //need to make linear index ids now
    //a variable for the index offsets
    unsigned int nodeIndexOffset = 0;
    for(unsigned int i = 0; i < numNodes; i++){
        for(unsigned int j = 0; j < tempNewNodes[i].size(); j++){
            tempNewNodes[i][j].nodeIndex += nodeIndexOffset;
            for(unsigned int k = 0; k < tempNewNodes[i][j].neighborIndex.size(); k++){
                tempNewNodes[i][j].neighborIndex[k] += nodeIndexOffset;
            }
        }
        nodeIndexOffset += tempNewNodes[i].size();
    }

    updateNeighbors(nodes,&tempNewNodes);

    // printf("finsihed updating neighbors\n");


    //make a new linear array of nodes
    std::vector<struct Node> nodeVec;
    for(unsigned int i = 0; i < tempNewNodes.size(); i++){
        nodeVec.insert(nodeVec.end(), tempNewNodes[i].begin(), tempNewNodes[i].end());
        // for(unsigned int j = 0; j < tempNewNodes[i].size(); j++){
        //     nodeVec.push_back(tempNewNodes[i][j]);
        // }
    }

    updateNodeCalcs(&nodeVec, nodeVec.size());

    // printf("NumNodes: %u, TotalCalcs: %llu\n", (unsigned int)nodeVec.size(), totalNodeCalcs(nodeVec, nodeVec.size()));

    *newNodes = nodeVec;

#if DEVICE_BUILD
    // hipFree(d_binNumber);
    // hipFree(d_RP);
    // free(allBinNumber);
#endif

    double time2 = omp_get_wtime();

    *nodePerSecond = numNodes/(time2-time1);

    return (unsigned int)nodeVec.size();

}


struct Node newNode(unsigned int numNodePoints, //number of points to go into the node
                    unsigned int * nodePoints, // the start of the points that will go into the node
                    struct Node parent, //the parent node
                    unsigned int binNumber,//the bin number of the node
                    unsigned int nodeNumber){ //the index number of the node

    struct Node newNode;
    newNode.nodeIndex = nodeNumber;
    newNode.numNodePoints = numNodePoints;
    // newNode.binNumbers = parent.binNumbers;
    for(unsigned int i = 0; i < parent.binNumbers.size();i++){
        newNode.binNumbers.push_back(parent.binNumbers[i]);
    }
    newNode.binNumbers.push_back(binNumber);
    newNode.neighborIndex.push_back(nodeNumber);
    // newNode.nodePoints.insert(newNode.nodePoints.begin(), &nodePoints[0], &nodePoints[numNodePoints-1] ); //double check this
    newNode.nodePoints.resize(numNodePoints);
    for(unsigned int i = 0; i < numNodePoints; i++){
        newNode.nodePoints[i] = nodePoints[i];
    }
    return newNode;
}; 

struct Node newNode(unsigned int numNodePoints, //number of points to go into the node
                    unsigned int * nodePoints, // the start of the points that will go into the node
                    unsigned int binNumber,//the bin number of the node
                    unsigned int nodeNumber){ //the index number of the node

    struct Node newNode;
    newNode.nodeIndex = nodeNumber;
    newNode.numNodePoints = numNodePoints;
    newNode.binNumbers.push_back(binNumber);
    newNode.neighborIndex.push_back(nodeNumber);
    // newNode.nodePoints.insert(newNode.nodePoints.begin(), &nodePoints[0], &nodePoints[numNodePoints-1]); //double check this
    newNode.nodePoints.resize(numNodePoints);
    for(unsigned int i = 0; i < numNodePoints; i++){
        newNode.nodePoints[i] = nodePoints[i];
    }
    return newNode;
};

void updateNodeCalcs(std::vector<struct Node> * nodes,
                     unsigned int numNodes){

    bool verboseNodeInfo = false;
    #pragma omp parallel for
    for(unsigned int i = 0; i < numNodes; i++){
        if(verboseNodeInfo) printf("Node %d has:\n",i);
        unsigned long long int numNeighboringPoints = 0;
        if(verboseNodeInfo) printf("    %u points\n", (*nodes)[i].numNodePoints);
        if(verboseNodeInfo) printf("    %lu neighbors\n", (*nodes)[i].neighborIndex.size());
        for(unsigned int j = 0; j < (*nodes)[i].neighborIndex.size(); j++){
            if(verboseNodeInfo) printf("    neighbors bin: %d with numPoints: %u\n", (*nodes)[i].neighborIndex[j],(*nodes)[(*nodes)[i].neighborIndex[j]].numNodePoints);
            numNeighboringPoints += (unsigned long long int)(*nodes)[(*nodes)[i].neighborIndex[j]].numNodePoints;
        }
        (*nodes)[i].numCalcs = (unsigned long long int)numNeighboringPoints*(*nodes)[i].numNodePoints;
        if(verboseNodeInfo) printf("    %llu total calcs to make\n", (*nodes)[i].numCalcs);
    }

}

unsigned long long totalNodeCalcs(std::vector<struct Node> nodes, unsigned int numNodes){
    unsigned long long totalCalcs = 0;
    
    for(unsigned int i = 0; i < nodes.size(); i++){
        
        totalCalcs += nodes[i].numCalcs;
    }

    return totalCalcs;
}

unsigned long long nodeSumSqrs(std::vector<struct Node> nodes, unsigned int numNodes){
    unsigned long long sumSqrs = 0;
    
    for(unsigned int i = 0; i < nodes.size(); i++){
        sumSqrs += nodes[i].numNodePoints*nodes[i].numNodePoints;
    }

    return sumSqrs;
}


void updateNeighbors(std::vector<struct Node> nodes, std::vector<std::vector<struct Node>> * newNodes){
        // go through the old node list and compare bins to adjacent splits of old nodes
    #pragma omp parallel for //num_threads(64/RPPERLAYER)
    for(unsigned int i = 0; i < nodes.size(); i++){ //for each old node
        for(unsigned int j = 0; j < (*newNodes)[i].size(); j++){ //go through each split off node
            // bin number that we are looking for adjacents to
            unsigned int nodeBinNumber = (*newNodes)[i][j].binNumbers.back();
            for(unsigned int k = 1; k < nodes[i].neighborIndex.size(); k++){ //and check the neighbors
                //neighbor to check
                unsigned int neighborNodesIndex = nodes[i].neighborIndex[k]; // this will also give the index of the vector of split nodes
                if(neighborNodesIndex > (*newNodes).size()-1) printf("ERROR: neighbor index: %u max: %u\n", neighborNodesIndex,(*newNodes).size()-1 );
                //go through each neighbors split nodes
                for(unsigned int l = 0; l < (*newNodes)[neighborNodesIndex].size(); l++){
                    if((*newNodes)[neighborNodesIndex][l].split == false ||
                       (*newNodes)[i][j].split == false) {
                            (*newNodes)[i][j].neighborIndex.push_back((*newNodes)[neighborNodesIndex][l].nodeIndex);
                    }else{
                        unsigned int checkBin = (*newNodes)[neighborNodesIndex][l].binNumbers.back();
                        if( checkBin + 1 == nodeBinNumber || 
                            checkBin - 1 == nodeBinNumber ||
                            checkBin == nodeBinNumber){
                                (*newNodes)[i][j].neighborIndex.push_back((*newNodes)[neighborNodesIndex][l].nodeIndex);
                        }
                    }
                }
            }
        }
    }
}

unsigned long long nodeForce(std::vector<struct Node> * nodes, double epsilon, double * data, unsigned int dim, unsigned int numPoints){

    std::vector<unsigned int> PA;
    std::vector<unsigned int> PB;
    
    bool check1 = true;
    bool check2 = true;
    bool verboseNodeInfo = true;
    // #pragma omp parallel for
    if(verboseNodeInfo)printf("\n************************************************\n");
    if(verboseNodeInfo)printf("Number of Nodes: %lu\n", (*nodes).size());
    for(unsigned int i = 0; i < (*nodes).size();i++){
        if(verboseNodeInfo) printf("Node: %u\n  numNeighbors: %lu\n", i, (*nodes)[i].neighborIndex.size());
        unsigned long long sum = 0;
        for(unsigned int j = 0; j < (*nodes)[i].neighborIndex.size();j++){
            if(verboseNodeInfo) printf("    neighbor at index: %u\n", (*nodes)[i].neighborIndex[j]);
            if(verboseNodeInfo) printf("        numPoints: %u\n", (*nodes)[(*nodes)[i].neighborIndex[j]].numNodePoints);
            if(verboseNodeInfo){
                printf("        Bin Numbers: |");
                for(unsigned int h = 0; h <  (*nodes)[(*nodes)[i].neighborIndex[j]].binNumbers.size(); h++){
                    printf(" %u |", (*nodes)[(*nodes)[i].neighborIndex[j]].binNumbers[h]);
                }
                printf("\n");
            }
            for(unsigned int k = 0; k < (*nodes)[i].numNodePoints; k++){
                for(unsigned int l = 0; l <(*nodes)[(*nodes)[i].neighborIndex[j]].numNodePoints; l++){
                    unsigned int a = (*nodes)[i].nodePoints[k];
                    unsigned int b = (*nodes)[(*nodes)[i].neighborIndex[j]].nodePoints[l];
                    if(a == 50 && check1) {
                        printf("                 point 50 Node Number: %u\n", i);
                        check1 = false;
                    }
                    if(a == 53 && check2) {
                        printf("                 point 53 Node Number: %u\n", i);
                        check2 = false;
                    }
                    if(verboseNodeInfo) if(b == a && j != 0) printf("ERROR:%u\n",a);
                    double running = 0;
                    for(unsigned int d = 0; d < dim; d++){
                        running += pow(data[a*dim + d] - data[b*dim + d], 2);
                    }
                    if(running <= epsilon*epsilon){
                        sum++;
                        PA.push_back(a);
                        PB.push_back(b);
                    }
                }

            }
        }
        (*nodes)[i].numResults = sum;
    }

    unsigned long long total = 0;
    for(unsigned int i = 0; i < (*nodes).size(); i++){
        total += (*nodes)[i].numResults;
    }
    // if(verboseNodeInfo)printf("TOTAL PAIRS: %llu\n", total);

    std::vector<unsigned int> BA;
    std::vector<unsigned int> BB;

    unsigned int brute_count = 0;
	omp_lock_t brute;
	omp_init_lock(&brute);

	#pragma omp parallel for
	for(unsigned int i = 0; i < numPoints; i++)
	{
		for (unsigned int j = 0; j < numPoints; j++)
		{
		double distance = 0;
			for (unsigned int k = 0; k < dim; k++)
			{
				if(distance > epsilon*epsilon)
				{
					break;
				} else {
					double a1 = data[i*dim + k];
					double a2 = data[j*dim + k];
					distance += (a1-a2)*(a1-a2);
				}
				}
				if(distance <= epsilon*epsilon){
					omp_set_lock(&brute);
					brute_count++;
                    BA.push_back(i);
                    BB.push_back(j);
					omp_unset_lock(&brute);
				}
		}
	}

    // printf("BrUTe count: %u", brute_count);

    std::vector< std::pair<unsigned int,unsigned int>> pairsB;

    for(unsigned int i = 0; i < BA.size();i++){
        pairsB.push_back(std::make_pair(BA[i],BB[i]));
    }

    std::sort(pairsB.begin(), pairsB.end(), compPair);

    std::vector< std::pair<unsigned int,unsigned int>> pairs;

    for(unsigned int i = 0; i < PA.size();i++){
        pairs.push_back(std::make_pair(PA[i],PB[i]));
    }

    std::sort(pairs.begin(), pairs.end(), compPair);

    pairs.erase(std::unique(pairs.begin(), pairs.end()), pairs.end());
    
    std::vector< std::pair<unsigned int,unsigned int>> missing;

    // std::set_difference(pairsB.begin(), pairsB.end(), pairs.begin(), pairs.end(), std::inserter(missing, missing.begin()));
    unsigned int counter = 0;
    for(unsigned int i = 0; i < pairsB.size();i++){
        if(pairsB[i].first == pairs[counter].first && pairsB[i].second == pairs[counter].second){
            counter++;
        }else{
            missing.push_back(pairsB[i]);
        }
    }

    printf("Missing %lu pairs:\n", missing.size());
    for(unsigned int i = 0; i < missing.size(); i++){
        printf("(%u,%u),",missing[i].first, missing[i].second);
    }

    printf("\nTotal NodeFORCE results Set Size: %llu , unique pairs: %lu\n", total, pairs.size());

    if(verboseNodeInfo)printf("\n************************************************\n");


    return total;
}

std::vector<std::vector<struct Node>> genSubGraphs(std::vector<struct Node> inNodes){


    //pull out all of the sub graphs
    // printf("Generating subs\n");
    std::vector<struct Node> nodes;
    nodes.insert(nodes.begin(), inNodes.begin(), inNodes.end());
    std::vector<std::vector<struct Node>> subGraphs;
    std::stack<unsigned int> stack;
    for(unsigned int i = 0; i < nodes.size(); i++){
        if(nodes[i].visited == false){
            std::vector<struct Node> newGraph;
            newGraph.push_back(nodes[i]);
            nodes[i].visited = true;
            stack.push(nodes[i].nodeIndex);

            while(stack.size() > 0){
                unsigned int current = stack.top(); 
                stack.pop();
                for(unsigned int j = 0; j < nodes[current].neighborIndex.size();j++){
                    if(nodes[nodes[current].neighborIndex[j]].visited == false){
                        stack.push(nodes[current].neighborIndex[j]);
                        newGraph.push_back(nodes[nodes[current].neighborIndex[j]]);
                        nodes[nodes[current].neighborIndex[j]].visited = true;
                    }
                }
                
            }

            subGraphs.push_back(newGraph);
        }
         
    }

    //fix neighbor pointers to be local to sub graphs
    // unsigned int offsetCounter = 0;
    for(unsigned int i = 0; i < subGraphs.size();i++){
        // printf("\n#####################\nOffset counter for sub %u: %u  :   : %u nodes in sub\n", i, offsetCounter, subGraphs[i].size());
        for(unsigned int j = 0; j < subGraphs[i].size(); j++){
            nodes[subGraphs[i][j].nodeIndex].nodeIndex = j;
            subGraphs[i][j].nodeIndex = j;
            // printf("sub/node: %u::%u node index: %u\n   bins:", i,j, subGraphs[i][j].nodeIndex);
            // for(unsigned int b = 0; b < subGraphs[i][j].binNumbers.size(); b++){
            //     printf("%u, ",subGraphs[i][j].binNumbers[b] );
            // }
            // printf("\n  -------- \n");

            // subGraphs[i][j].neighborIndex.clear();
        }

        // for(unsigned int j = 0; j < subGraphs[i].size(); j++){
        //     unsigned int nodeBinNumber = subGraphs[i][j].binNumbers.back();
        //     for(unsigned int k = 0; k < subGraphs[i].size(); k++){
        //         if(subGraphs[i][k].split == false ||
        //                 subGraphs[i][j].split == false) {
        //                     subGraphs[i][j].neighborIndex.push_back(subGraphs[i][k].nodeIndex);
        //         }else{
        //             unsigned int checkBin = subGraphs[i][k].binNumbers.back();
        //             if( checkBin + 1 == nodeBinNumber || 
        //                 checkBin - 1 == nodeBinNumber ||
        //                 checkBin == nodeBinNumber){
        //                     subGraphs[i][j].neighborIndex.push_back(subGraphs[i][k].nodeIndex);
        //             }
        //         }
        //     }
        for(unsigned int j = 0; j < subGraphs[i].size(); j++){
            for(unsigned int k = 0; k < subGraphs[i][j].neighborIndex.size(); k++){
                
                subGraphs[i][j].neighborIndex[k] = nodes[subGraphs[i][j].neighborIndex[k]].nodeIndex;

                // printf("    sub: %u, node: %u at index %u: %u\n     bins:", i, j,k, subGraphs[i][j].neighborIndex[k]);
                // for(unsigned int b = 0; b < subGraphs[i][j].binNumbers.size(); b++){
                //     printf("%u, ",subGraphs[i][ subGraphs[i][j].neighborIndex[k] ].binNumbers[b] );
                // }
                // printf("\n");
                
            }
        }
        
        
        // offsetCounter += subGraphs[i].size();
    }
    
    printf("Num SubGraphs: %u\n", subGraphs.size());

    return subGraphs;

}