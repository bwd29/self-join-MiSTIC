#include "hip/hip_runtime.h"

#include "include/nodes.cuh"





//function to build up net of nodes
unsigned int buildNodeNet(double * data,
                 unsigned int dim,
                 unsigned int numPoints,
                 unsigned int numRP,
                 unsigned int * pointArray,
                 double epsilon,
                 struct Node ** nodes){

    // generate some reference points
    
    struct Node * newNodes;
    unsigned int numNodes;
    
    // need to go through each reference point
    for(unsigned int i = 0; i < numRP; i++){ 
        double * RPArray = createRPArray(data, numRP, dim, numPoints);
        struct Node * layerNodes;
        unsigned long long int lowestDistCalcs = ULLONG_MAX;
        unsigned int bestRP = 0;
        for(unsigned int j = 0; j < RPPERLAYER; j++){
            // need to compare num dist calcs for different potental RP
            struct Node * tempNodes;
            unsigned int tempNumNodes;
            if(i == 0){tempNumNodes = initNodes(data, dim, numPoints, epsilon, &RPArray[j*dim], pointArray, &tempNodes);}
            else{tempNumNodes = splitNodes(&RPArray[j*dim], newNodes, numNodes, epsilon, data, dim, numPoints, &tempNodes);}
            unsigned long long numCalcs = totalNodeCalcs(tempNodes, tempNumNodes);
            unsigned long long sumSqrs = nodeSumSqrs(tempNodes, tempNumNodes);
            printf("Layer %d for RP %d has Nodes: %u with calcs: %llu , and sumSQRs: %llu\n", i, j, tempNumNodes, numCalcs, sumSqrs);

            if(numCalcs < lowestDistCalcs){
                lowestDistCalcs = numCalcs;
                bestRP = j;
                layerNodes = tempNodes;
                numNodes = tempNumNodes;
            }
            
        }

        printf("Layer %d Selecting RP %d with Nodes: %u and calcs: %llu\n", i, bestRP, numNodes, lowestDistCalcs);

        newNodes = layerNodes;
    }

    unsigned long long numCalcs = totalNodeCalcs(&newNodes[0], numNodes);
    unsigned long long sumSqrs = nodeSumSqrs(&newNodes[0], numNodes);

    printf("Final graph has %u nodes with: %llu calcs and sumSqrs: %llu", numNodes, numCalcs, sumSqrs);

    *nodes = &newNodes[0];

    //rearange the pointArray
    unsigned int counter = 0;
    for(unsigned int i = 0; i < numNodes; i++){
        newNodes[i].pointOffset = counter;
        for(unsigned int j = 0; j < newNodes[i].numNodePoints; j++){
            pointArray[counter] = newNodes[i].nodePoints[j];
            counter++;
        }
    }


    return numNodes;

}

unsigned int initNodes(double * data,
                        unsigned int dim,
                        unsigned int numPoints,
                        double epsilon,
                        double * RP,
                        unsigned int * pointArray,
                        struct Node ** nodes){



    std::vector<struct Node> newNodes;

    //make the first set of nodes
    unsigned int * binNumber = (unsigned int * )malloc(sizeof(unsigned int)*numPoints);
    for(unsigned int i = 0; i < numPoints; i++){
        //get distance of each point in the node to the reference point
        binNumber[i] = floor( euclideanDistance(&data[i*dim],dim,RP) / epsilon);
    }

    // sort the node points based on their bin numbers
    thrust::sort_by_key(thrust::host, &binNumber[0], &binNumber[numPoints-1], &pointArray[0]);


    //if all the points are in the same bin
    if(binNumber[0] == binNumber[numPoints-1]){

        newNodes.push_back(newNode(numPoints, pointArray, binNumber[0], 0));
        *nodes = &newNodes[0];
        //free temp memory
        free(binNumber);

        //go to the next node
        return 1;
    }

    //go through and make nodes

    //variable to keep track of last bin end
    unsigned int tempBinPointer = 0;

    //variable to count new nodes
    unsigned int numNewNodes = 0;

    //scan through and create a new node for each non-empty bin
    for(unsigned int i = 0; i < numPoints; i++){

        //check if need to make a new node
        if(i == numPoints-1 || binNumber[i] != binNumber[i+1]){
            //push back the new node onto the temporary vector of nodes
            newNodes.push_back( newNode(i-tempBinPointer, pointArray, binNumber[i], numNewNodes ) );
            tempBinPointer = i;
            numNewNodes++;
        }
    }

    //create the connections
            //now that the split nodes exist, modify neighbor values
    //special case for the first
    if(newNodes[0].binNumbers.back() == newNodes[1].binNumbers.back() - 1 ){ //already know at least 2 nodes in list
        newNodes[0].neighborIndex.push_back(1);
    } 

    // handle all middle nodes
    for(unsigned int i = 1; i < numNewNodes-1; i++){
        //check if lower bin is one away
        if(newNodes[i].binNumbers.back() == newNodes[i-1].binNumbers.back() + 1){
            newNodes[i].neighborIndex.push_back(i-1);
        }
        //check if upper bin is one away
        if(newNodes[i].binNumbers.back() == newNodes[i+1].binNumbers.back() - 1){
            newNodes[i].neighborIndex.push_back(i+1);
        }
    }

    //special case for last node
    if(newNodes[numNewNodes-1].binNumbers.back() == newNodes[numNewNodes-2].binNumbers.back() + 1 ){ //already know at least 2 nodes in list
        newNodes[numNewNodes-1].neighborIndex.push_back(numNewNodes-2);
    }


    updateNodeCalcs(&newNodes[0], newNodes.size());
    //assign the vector to the return
    *nodes = &newNodes[0];

    free(binNumber);

    return numNewNodes;

}

//splits a node based on a reference point and return the number of new nodes
unsigned int splitNodes(double * RP, //the reference point used for the split
                    struct Node* nodes,// the array of nodes
                    unsigned int numNodes,//the number of nodes
                    double epsilon, //the distance threshold of the search
                    double * data, //the dataset
                    unsigned int dim,//the number of dimensions of the data
                    unsigned int numPoints,// number of points in the dataset
                    struct Node ** newNodes){  // pointer for returning the new nodes
    

    printf("Start split node\n");
    //need to keep track of all of the new split nodes
    std::vector<std::vector<struct Node>> tempNewNodes;
    tempNewNodes.resize(numNodes);
   
    printf("allocate vector node\n");


    // go through each node and split
    for(unsigned int i = 0; i < numNodes; i++){


        //temp array to hold each points new bin number
        unsigned int * binNumber = (unsigned int * )malloc(sizeof(unsigned int)*nodes[i].numNodePoints);


        // break nodes into new nodes
        for(unsigned int j = 0; j < nodes[i].numNodePoints; j++){
            //get distance of each point in the node to the reference point
            binNumber[j] = floor( euclideanDistance(&data[j*dim],dim,RP) / epsilon);
        }


        // sort the node points based on their bin numbers
        thrust::sort_by_key(thrust::host, &binNumber[0], &binNumber[nodes[i].numNodePoints-1], &nodes[i].nodePoints[0]);

        //if all the points are in the same bin
        if(binNumber[0] == binNumber[nodes[i].numNodePoints-1]){

            //add the bin number
            nodes[i].binNumbers.push_back(binNumber[0]);

            //free temp memory
            free(binNumber);

            //go to the next node
            continue;
        }



        //temp vector to hold new nodes
        std::vector<struct Node> tempNodes;


        //variable to keep track of last bin end
        unsigned int tempBinPointer = 0;

        //variable to count new nodes
        unsigned int numNewNodes = 0;

        //scan through and create a new node for each non-empty bin
        for(unsigned int j = 0; j < nodes[i].numNodePoints; j++){
            
            //check if need to make a new node
            if(j == nodes[i].numNodePoints-1 || binNumber[j] != binNumber[j+1]){
                //push back the new node onto the temporary vector of nodes
                tempNodes.push_back( newNode(j-tempBinPointer, &(nodes[j].nodePoints[tempBinPointer]), nodes[i], binNumber[j], numNewNodes ) );
                tempBinPointer = j;
                numNewNodes++;
            }
        }

        //now that the split nodes exist, modify neighbor values
        //special case for the first
        if(tempNodes[0].binNumbers.back() == tempNodes[1].binNumbers.back() - 1 ){ //already know at least 2 nodes in list
            tempNodes[0].neighborIndex.push_back(1);
        } 

        // handle all middle nodes
        for(unsigned int j = 1; j < numNewNodes-1; j++){
            //check if lower bin is one away
            if(tempNodes[j].binNumbers.back() == tempNodes[j-1].binNumbers.back() + 1){
                tempNodes[j].neighborIndex.push_back(j-1);
            }
            //check if upper bin is one away
            if(tempNodes[j].binNumbers.back() == tempNodes[j+1].binNumbers.back() - 1){
                tempNodes[j].neighborIndex.push_back(j+1);
            }
        }

        //special case for last node
        if(tempNodes[numNewNodes-1].binNumbers.back() == tempNodes[numNewNodes-2].binNumbers.back() + 1 ){ //already know at least 2 nodes in list
            tempNodes[numNewNodes-1].neighborIndex.push_back(numNewNodes-2);
        } 

        
        //copy the temp nodes back into the larger vector of nodes
        tempNewNodes[i]  = tempNodes;
        free(binNumber);
    }

    ////////////////////////////////////////////////////////////////
    // All nodes in the temp nodes now point to their neighbors within the smaller array
    // the neighbor index values will have to be updated when merging with the other vectors
    /////////////////////////////////////////////////////////////

    //need to make linear index ids now
    //a variable for the index offsets
    unsigned int nodeIndexOffset = 0;
    for(unsigned int i = 0; i < numNodes; i++){
        for(unsigned int j = 0; j < tempNewNodes[i].size(); j++){
            tempNewNodes[i][j].nodeIndex += nodeIndexOffset;
            for(unsigned int k = 0; k < tempNewNodes[i][j].neighborIndex.size(); k++){
                tempNewNodes[i][j].neighborIndex[k] += nodeIndexOffset;
            }
        }
        nodeIndexOffset += tempNewNodes[i].size();
    }

    // go through the old node list and compare bins to adgacent splits of old nodes

    for(unsigned int i = 0; i < numNodes; i++){ //for eacch old node
        for(unsigned int j = 0; j < tempNewNodes[i].size(); j++){ //go through each split off node
            // bin number that we are lloking for adjacents to
            unsigned int nodeBinNumber = tempNewNodes[i][j].binNumbers.back();
            for(unsigned int k = 0; k < nodes[i].neighborIndex.size(); k++){ //and check the neighbors
                //neighbor to check
                unsigned int neighborNodesIndex = nodes[i].neighborIndex[k]; // this will also give the index of the vector of split nodes
                
                //go through each neighbors split nodes
                for(unsigned int l = 0; l < tempNewNodes[neighborNodesIndex].size(); l++){
                    unsigned int checkBin = tempNewNodes[neighborNodesIndex][l].binNumbers.back();
                    if(checkBin + 1 == nodeBinNumber || 
                        checkBin - 1 == nodeBinNumber ||
                        checkBin == nodeBinNumber){
                        tempNewNodes[i][j].neighborIndex.push_back(tempNewNodes[neighborNodesIndex][l].nodeIndex);
                    }
                }
            }
        }
    }


    //make a new linear array of nodes
    std::vector<struct Node> nodeVec;
    for(unsigned int i = 0; i < numNodes; i++){
        nodeVec.insert(nodeVec.end(), tempNewNodes[i].begin(), tempNewNodes[i].end());
    }

    updateNodeCalcs(&nodeVec[0], nodeVec.size());

    printf("NumNodes: %u, TotalCalcs: %llu", (unsigned int)nodeVec.size(), totalNodeCalcs(&nodeVec[0], nodeVec.size()));

    *newNodes = &nodeVec[0];

    return (unsigned int)nodeVec.size();

}


struct Node newNode(unsigned int numNodePoints, //number of points to go into the node
                    unsigned int * nodePoints, // the start of the points that will go into the node
                    struct Node parent, //the parent node
                    unsigned int binNumber,//the bin number of the node
                    unsigned int nodeNumber){ //the index number of the node

    struct Node newNode;
    newNode.nodeIndex = nodeNumber;
    newNode.numNodePoints = numNodePoints;
    newNode.binNumbers = parent.binNumbers;
    newNode.binNumbers.push_back(binNumber);
    newNode.nodePoints.insert(newNode.nodePoints.begin(), &nodePoints[0], &nodePoints[numNodePoints-1] ); //double check this

    return newNode;
};

struct Node newNode(unsigned int numNodePoints, //number of points to go into the node
                    unsigned int * nodePoints, // the start of the points that will go into the node
                    unsigned int binNumber,//the bin number of the node
                    unsigned int nodeNumber){ //the index number of the node

    struct Node newNode;
    newNode.nodeIndex = nodeNumber;
    newNode.numNodePoints = numNodePoints;
    newNode.binNumbers.push_back(binNumber);
    newNode.nodePoints.insert(newNode.nodePoints.begin(), &nodePoints[0], &nodePoints[numNodePoints-1]); //double check this

    return newNode;
};

void updateNodeCalcs(struct Node * nodes,
                     unsigned int numNodes){

    for(unsigned int i = 0; i < numNodes; i++){
        unsigned long long int numNeighboringPoints = 0;
        for(unsigned int j = 0; j < nodes[i].neighborIndex.size(); j++){
            numNeighboringPoints += nodes[nodes[i].neighborIndex[j]].numNodePoints;
        }
        nodes[i].numCalcs = numNeighboringPoints*nodes[i].numNodePoints;
    }

}

unsigned long long totalNodeCalcs(struct Node * nodes, unsigned int numNodes){
    unsigned long long totalCalcs = 0;
    
    for(unsigned int i = 0; i < numNodes; i++){
        totalCalcs += nodes[i].numCalcs;
    }

    return totalCalcs;
}

unsigned long long nodeSumSqrs(struct Node * nodes, unsigned int numNodes){
    unsigned long long totalCalcs = 0;
    
    for(unsigned int i = 0; i < numNodes; i++){
        totalCalcs += nodes[i].numNodePoints*nodes[i].numNodePoints;
    }

    return totalCalcs;
}